#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <thread>
#include <future>
#include <string>
#include "gensupermer.hpp"

#include "utilities.hpp"
using namespace std;

__device__ __constant__ static const unsigned char d_basemap[256] = {
    255, 255, 255, 255, 255, 255, 255, 255, // 0..7
    255, 255, 255, 255, 255, 255, 255, 255, // 8..15
    255, 255, 255, 255, 255, 255, 255, 255, // 16..23
    255, 255, 255, 255, 255, 255, 255, 255, // 24..31
    255, 255, 255, 255, 255, 255, 255, 255, // 32..39
    255, 255, 255, 255, 255, 255, 255, 255, // 40..47
    255, 255, 255, 255, 255, 255, 255, 255, // 48..55
    255, 255, 255, 255, 255, 255, 255, 255, // 56..63
    255, 0, 255, 1, 255, 255, 255, 2, // 64..71
    255, 255, 255, 255, 255, 255, 255, 255, // 72..79
    255, 255, 255, 255, 3, 255, 255, 255, // 80..87
    255, 255, 255, 255, 255, 255, 255, 255, // 88..95
    255, 0, 255, 1, 255, 255, 255, 2, // 96..103
    255, 255, 255, 255, 255, 255, 255, 255, // 104..111
    255, 255, 255, 255, 3, 255, 255, 255, // 112..119
    255, 255, 255, 255, 255, 255, 255, 255, // 120..127
    255, 255, 255, 255, 255, 255, 255, 255, // 128..135
    255, 255, 255, 255, 255, 255, 255, 255, // 136..143
    255, 255, 255, 255, 255, 255, 255, 255, // 144..151
    255, 255, 255, 255, 255, 255, 255, 255, // 152..159
    255, 255, 255, 255, 255, 255, 255, 255, // 160..167
    255, 255, 255, 255, 255, 255, 255, 255, // 168..175
    255, 255, 255, 255, 255, 255, 255, 255, // 176..183
    255, 255, 255, 255, 255, 255, 255, 255, // 184..191
    255, 255, 255, 255, 255, 255, 255, 255, // 192..199
    255, 255, 255, 255, 255, 255, 255, 255, // 200..207
    255, 255, 255, 255, 255, 255, 255, 255, // 208..215
    255, 255, 255, 255, 255, 255, 255, 255, // 216..223
    255, 255, 255, 255, 255, 255, 255, 255, // 224..231
    255, 255, 255, 255, 255, 255, 255, 255, // 232..239
    255, 255, 255, 255, 255, 255, 255, 255, // 240..247
    255, 255, 255, 255, 255, 255, 255, 255  // 248..255
};

typedef struct {
    _in_ T_read_count cur_batch_size;    //number of reads
    
    // Raw reads
    _in_ _out_ char *reads;              //read_CSR 
    _in_ T_CSR_capacity *reads_offs;     //read_CSR_offset 
    _in_ _out_ T_read_len *read_len;      //length of each read
    
    // Minimizers
    _out_ _tmp_ T_minimizer *minimizers;  //the minimizer array
    _out_ T_read_len *supermer_offs;     //the supermer offset array
} T_GPU_data;

/*
 * [INPUT]  data.reads in [(Read#0), (Read#1)...]
 * [OUTPUT] data.minimizers in [(Read#0)[mm1, mm?, mm?, ...], (Read#1)...]
 */
__global__ void GPU_GenMinimizer(_in_ _out_ T_GPU_data data, int K_kmer, int P_minimizer) {

    // Fill in the GPU_GenMinimizer function here

    return;
}


/* [INPUT]  data.minimizers in [[mm1, mm1, mm2, mm3, ...], ...]
 * [OUTPUT] data.supermer_offs in [[0, 2, 3, ...], ...]
 */
__global__ void GPU_GenSKM(_in_ _out_ T_GPU_data data, int K_kmer, int P_minimizer) {

    // Fill in the GPU_GenSKM function here

    return;
}


void GenerateSupermer_GPU(vector<string> &reads, int K, int P, vector<string> &all_supermers, int NUM_BLOCKS_PER_GRID, int NUM_THREADS_PER_BLOCK) {


    T_GPU_data d_batch_data, h_batch_data;
    T_read_count cur_batch_size;

    CSR<char, int> csr_reads(true);
        
    // convert raw read from vector to csr:
    int name_idx_in_batch = 0;
    for(string read: reads) {
        csr_reads.append(read.c_str(), read.length(),name_idx_in_batch);
        name_idx_in_batch++;
    }
    cur_batch_size = reads.size();
    // malloc and memcpy from host to device
    h_batch_data.reads = new char[csr_reads.size()];
    h_batch_data.reads_offs = csr_reads.get_raw_offs();
    h_batch_data.read_len = new T_read_len[csr_reads.items()];
    h_batch_data.minimizers = new T_minimizer[csr_reads.size()];
    h_batch_data.supermer_offs = new T_read_len[csr_reads.size()];
    for (int i=0; i<name_idx_in_batch-1; i++) {
        h_batch_data.read_len[i] = csr_reads.get_raw_offs()[i+1] - csr_reads.get_raw_offs()[i];
    }
    h_batch_data.read_len[name_idx_in_batch-1] = csr_reads.size() - csr_reads.get_raw_offs()[name_idx_in_batch-1];

    // [Data H2D]
    // gpu malloc
    GPUErrChk(hipMalloc((void**) &(d_batch_data.reads), sizeof(char) * csr_reads.size()));
    GPUErrChk(hipMalloc((void**) &(d_batch_data.reads_offs), sizeof(size_t) * (csr_reads.items()+1)));
    d_batch_data.cur_batch_size = cur_batch_size;
    #ifdef DEBUG
    // cerr << "(GPU"<<GPU_ID<<"): batch_size = " << cur_batch_size << ", bases = " << csr_reads.size() << endl;
    // cerr << " <G" << GPU_ID << "> " << cur_batch_size << "|" << csr_reads.size();
    #endif
    GPUErrChk(hipMalloc((void**) &(d_batch_data.read_len), sizeof(T_read_len) * csr_reads.items()));
    GPUErrChk(hipMalloc((void**) &(d_batch_data.minimizers), sizeof(T_minimizer) * csr_reads.size()));
    GPUErrChk(hipMalloc((void**) &(d_batch_data.supermer_offs), sizeof(T_read_len) * csr_reads.size()));
    // memcpy Host -> Device
    GPUErrChk(hipMemcpy(d_batch_data.reads, csr_reads.get_raw_data(), sizeof(char) * csr_reads.size(), hipMemcpyHostToDevice));
    GPUErrChk(hipMemcpy(d_batch_data.read_len, h_batch_data.read_len, sizeof(T_read_len) * (csr_reads.items()), hipMemcpyHostToDevice));
    GPUErrChk(hipMemcpy(d_batch_data.reads_offs, csr_reads.get_raw_offs(), sizeof(size_t) * (csr_reads.items()+1), hipMemcpyHostToDevice));

    // [Computing]
    GPU_GenMinimizer<<<NUM_BLOCKS_PER_GRID, NUM_THREADS_PER_BLOCK/*, 0, hip_stream*/>>>(d_batch_data, K, P);
    GPU_GenSKM<<<NUM_BLOCKS_PER_GRID, NUM_THREADS_PER_BLOCK/*, 0, hip_stream*/>>>(d_batch_data, K, P);

    hipDeviceSynchronize();

    // [Data D2H]
    
    GPUErrChk(hipMemcpy(h_batch_data.reads, d_batch_data.reads, sizeof(char) * (csr_reads.size()), hipMemcpyDeviceToHost));
    GPUErrChk(hipMemcpy(h_batch_data.read_len, d_batch_data.read_len, sizeof(T_read_len) * (csr_reads.items()), hipMemcpyDeviceToHost));
    GPUErrChk(hipMemcpy(h_batch_data.minimizers, d_batch_data.minimizers, sizeof(T_minimizer) * (csr_reads.size()), hipMemcpyDeviceToHost));
    GPUErrChk(hipMemcpy(h_batch_data.supermer_offs, d_batch_data.supermer_offs, sizeof(T_read_len) * (csr_reads.size()), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    // GPUErrChk(hipStreamSynchronize(hip_stream));
    GPUErrChk(hipFree(d_batch_data.reads));
    GPUErrChk(hipFree(d_batch_data.reads_offs));
    GPUErrChk(hipFree(d_batch_data.minimizers));
    GPUErrChk(hipFree(d_batch_data.read_len));
    GPUErrChk(hipFree(d_batch_data.supermer_offs));
    hipDeviceSynchronize();
    
    for (int i=0; i<name_idx_in_batch; i++) {
        int skm_idx = 1;
        T_read_len *skm_offs = &h_batch_data.supermer_offs[h_batch_data.reads_offs[i]];
        vector<string>  supermers_local_all;
        T_read_len len = h_batch_data.read_len[i];
        while (*(skm_offs+skm_idx-1) != len- K +1) {
            int skm_len = *(skm_offs+skm_idx) - *(skm_offs+skm_idx-1) + K-1;
            char* t = (char *) malloc(skm_len+1);
            memcpy(t, h_batch_data.reads +h_batch_data.reads_offs[i]+ *(skm_offs+skm_idx-1), skm_len);
            t[skm_len] = '\0';
            supermers_local_all.push_back(t);
            skm_idx++;
        }

        all_supermers.insert(all_supermers.end(), supermers_local_all.begin(), supermers_local_all.end());
    }
    

    delete [] h_batch_data.reads;
    delete [] h_batch_data.read_len;
    delete [] h_batch_data.minimizers;
    delete [] h_batch_data.supermer_offs;


    return;
}
